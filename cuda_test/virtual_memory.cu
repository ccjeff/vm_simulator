﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h> 

__device__ int vm_swap(int phyPage, int virtPage, VirtualMemory* vm) {
	(*vm->pagefault_num_ptr)++;
	for (int i = 0; i < vm->PAGESIZE; i++) {
		uchar tempBuffer;
		tempBuffer = vm->storage[virtPage*vm->PAGESIZE + i];
		vm->buffer[phyPage*vm->PAGESIZE + i] = tempBuffer;
	}
	return 0;
}

__device__ u32 leastUsed(VirtualMemory* vm) {
	int tempSmallest = vm->invert_page_table[vm->PAGE_ENTRIES];
	int tempAddr = 0;
	for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
		if (vm->invert_page_table[i + vm->PAGE_ENTRIES] < tempSmallest) {
			tempSmallest = vm->invert_page_table[i + vm->PAGE_ENTRIES];
			tempAddr = i;
		}
	}
	
	//vm->invert_page_table[tempAddr + vm->PAGE_ENTRIES] += 1024;
	//printf("least1 should be: %d\n", vm->invert_page_table[1024]);
	vm->invert_page_table[tempAddr + vm->PAGE_ENTRIES] = tempSmallest + vm->PAGE_ENTRIES;
	//printf("least used index is: %d\n", tempAddr);
	return (u32) tempAddr;
}


__device__ int getPhyAddr(int addr, VirtualMemory* vm) {
	int physicalPage = 0;
	int virtualPage = addr / vm->PAGESIZE;
	for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
		if ((vm->invert_page_table[i]) == (virtualPage)) { // todo: 这个地方应该是addr/page? 1024个page entry怎么存？这样判断virtual addr 肯定不对应
			physicalPage = (u32)i;
			return physicalPage;
		}
	}
	// if not in physical memory, then must be in swap space
	return 0xFFFFFFFF;
}

__device__ void init_invert_page_table(VirtualMemory *vm) {
	for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
		vm->invert_page_table[i] = 0x80000000; // invalid := MSB is 1
		// 1000_0000_0000_0000_0000_0000_0000_0000
		// 0000_0000_0000_0000_0001_1111_1111_1111 
		/*only 13 bits are required for storing the 128kb virtual address*/ 
		vm->invert_page_table[i + vm->PAGE_ENTRIES] = i;
	}
}

__device__ void vm_init(VirtualMemory *vm, uchar *buffer, uchar *storage,
                        u32 *invert_page_table, int *pagefault_num_ptr,
                        int PAGESIZE, int INVERT_PAGE_TABLE_SIZE,
                        int PHYSICAL_MEM_SIZE, int STORAGE_SIZE,
                        int PAGE_ENTRIES) {
	// init variables
	vm->buffer = buffer;
	// buffer is the physical memory
	vm->storage = storage;
	// virtual memory
	vm->invert_page_table = invert_page_table;
	vm->pagefault_num_ptr = pagefault_num_ptr;

	// init constants
	vm->PAGESIZE = PAGESIZE;
	vm->INVERT_PAGE_TABLE_SIZE = INVERT_PAGE_TABLE_SIZE;
	vm->PHYSICAL_MEM_SIZE = PHYSICAL_MEM_SIZE;
	vm->STORAGE_SIZE = STORAGE_SIZE;
	vm->PAGE_ENTRIES = PAGE_ENTRIES;

	// before first vm_write or vm_read
	init_invert_page_table(vm);
}

__device__ uchar vm_read(VirtualMemory *vm, u32 addr) {
	/* Complate vm_read function to read single element from data buffer */
	//todo 
	uchar toReturn;
	int pageIdx = addr / vm->PAGESIZE;  // virtualPage index
	int offsetIdx = addr % vm->PAGESIZE;

	int memAddr = getPhyAddr(addr, vm);
	if (memAddr == 0xFFFFFFFF) {
		// do swap
		u32 leastIdx = leastUsed(vm);
		vm_swap(leastIdx,pageIdx,vm);
		memAddr = leastIdx;
	}
	printf("inside read, char get is %c\n", vm->buffer[memAddr*vm->PAGESIZE + offsetIdx]);
	return vm->buffer[memAddr*vm->PAGESIZE+offsetIdx];
}

__device__ void vm_write(VirtualMemory *vm, u32 addr, uchar value) {
	/* Complete vm_write function to write value into data buffer */
	int pageIdx = addr/vm->PAGESIZE;
	int offsetIdx = addr%vm->PAGESIZE;
	int physicalPage = 0xFFFFFFFF;
	
	physicalPage = getPhyAddr(addr,vm);
	// unsuccessful get will return 0xFFFFFFFF, which means needs swapping (or needs to be initialized).
	printf("getPhyAddr success , addr is: %d \n",addr);

	// if the page has been found in the pagetable
	printf("the physicalPage is: %d \n",physicalPage);
	if (physicalPage != 0xFFFFFFFF) {
		printf("found\n");
		int phyAddr = physicalPage * vm->PAGESIZE + offsetIdx;
		vm->buffer[phyAddr] = value;
		vm->invert_page_table[physicalPage + vm->PAGE_ENTRIES]+=vm->PAGE_ENTRIES;
		printf("normal write done \n");
	}
	else {
		// do swap before write
		int leastIdx = leastUsed(vm);
		if (vm->invert_page_table[leastIdx] == 0x80000000) {
			vm->buffer[leastIdx] = value;
			printf("value1 is: %c\n",value);
			vm->invert_page_table[leastIdx] = addr;
		}
		else {
			printf("the least index for swapping is: %d\n",leastIdx);
			printf("swap\n");
			vm_swap(leastIdx, pageIdx, vm);
			vm->buffer[leastIdx] = value;
			printf("value2 is: %c\n", value);
			vm->invert_page_table[leastIdx] = addr;
		}
	}
}

__device__ void vm_snapshot(VirtualMemory *vm, uchar *results, int offset,
                            int input_size) {
	/* Complete snapshot function togther with vm_read to load elements from data
	* to result buffer */
	printf("inside snapshot");
	for (int i = 0; i < vm->STORAGE_SIZE; i++) {
		results[offset + i] = vm_read(vm, ((u32)i));
	}
}

